#include "hip/hip_runtime.h"
// cuFHE includes
#include "cufhe/include/cufhe_gpu.cuh"
#include "cufhe/include/bootstrap_gpu.cuh"
#include "cufhe/include/cufhe.h"
#include "cufhe/include/ntt_gpu/ntt.cuh"

#include <cstdio>
#include <netdb.h>
#include <netinet/in.h>
#include <unistd.h>
#include <cstdlib>
#include <cstring>
#include <sys/socket.h>
#include <sys/types.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sys/stat.h>

#define MAX 100
#define PORT 8080
#define SA struct sockaddr
#define NUM_SMS 16

using namespace cufhe;
using namespace std;

int st_ctr = 0;

bool exists(const string& name) {
  struct stat buffer;
  return (stat (name.c_str(), &buffer) == 0);
}

string gen_filename()
{
  srand(time(0));
  bool isTaken = 1;
  char* fileName = (char*) malloc(28);
  while (isTaken != 0) {
      strcpy(fileName, "");
      strcpy(fileName, "encrypted_data/ctxt");
      int num = (rand() % 8999) + 1000;
      char strNum[sizeof(num)];
      sprintf(strNum, "%d", num);
      strcat(fileName, strNum);
      strcat(fileName, ".data\0");
      string strFileName(fileName, 28);
      isTaken = exists(strFileName);
  }
  string strFileName(fileName, 28);
  return strFileName;
}

// Constant Encryptions
void NoiselessTrivial(Ctxt& result, Torus mu, int32_t n=500){
    const int32_t degree = n;

    for (int32_t i = 0; i < degree; ++i) result.lwe_sample_->a()[i] = 0;
    result.lwe_sample_->b() = mu;
}

void Constant(Ctxt& result, int32_t value) {
    static const Torus MU = ModSwitchToTorus(1, 8);
    NoiselessTrivial(result, value ? MU : -MU);
}

// GPU <-> CPU data transfers
void CtxtCopyH2D(const Ctxt& c, Stream st) {
  hipMemcpyAsync(c.lwe_sample_device_->data(),
                  c.lwe_sample_->data(),
                  c.lwe_sample_->SizeData(),
                  hipMemcpyHostToDevice,
                  st.st());
}
void CtxtCopyD2H(const Ctxt& c, Stream st) {
  hipMemcpyAsync(c.lwe_sample_->data(),
                  c.lwe_sample_device_->data(),
                  c.lwe_sample_->SizeData(),
                  hipMemcpyDeviceToHost,
                  st.st());
}

// Logic Functions

// 0
void e_and(Ctxt* result, const Ctxt* a, const Ctxt* b, const int nb_bits, Stream* st_list) {
    for (int i=0; i<nb_bits; i++) {
        And(result[i], a[i], b[i], st_list[i%NUM_SMS]);
    }
}
// 1
void e_nand(Ctxt* result, const Ctxt* a, const Ctxt* b, const int nb_bits, Stream* st_list) {
    for (int i=0; i<nb_bits; i++) {
        Nand(result[i], a[i], b[i], st_list[i%NUM_SMS]);
    }
}
// 2
void e_or(Ctxt* result, const Ctxt* a, const Ctxt* b, const int nb_bits, Stream* st_list) {
    for (int i=0; i<nb_bits; i++) {
        Or(result[i], a[i], b[i], st_list[i%NUM_SMS]);
    }
}
// 3
void e_nor(Ctxt* result, const Ctxt* a, const Ctxt* b, const int nb_bits, Stream* st_list) {
    for (int i=0; i<nb_bits; i++) {
        Nor(result[i], a[i], b[i], st_list[i%NUM_SMS]);
    }
}
// 4
void e_xor(Ctxt* result, const Ctxt* a, const Ctxt* b, const int nb_bits, Stream* st_list) {
    for (int i=0; i<nb_bits; i++) {
        Xor(result[i], a[i], b[i], st_list[i%NUM_SMS]);
    }
}
// 5
void e_xnor(Ctxt* result, const Ctxt* a, const Ctxt* b, const int nb_bits, Stream* st_list) {
    for (int i=0; i<nb_bits; i++) {
        Xnor(result[i], a[i], b[i], st_list[i%NUM_SMS]);
    }
}
// 9
void e_not(Ctxt* result, const Ctxt* a, const int nb_bits, Stream* st_list) {
    for (int i=0; i<nb_bits; i++) {
        Not(result[i], a[i], st_list[i%NUM_SMS]);
    }
}
// 10
void e_shl(Ctxt* result, Ctxt* temp, const Ctxt* a, const int shift_amt, const int nb_bits, Stream* st_list) {
  // Ctxt* temp = new Ctxt[nb_bits];

  for(int i = 0; i < nb_bits; i++) {
    Copy(result[i], a[i], st_list[i%NUM_SMS]);
  }

  for(int i = 0; i < shift_amt; i++) {
    for(int j = 1; j < nb_bits; j++) {
      Copy(temp[j], result[j-1], st_list[i%NUM_SMS]);
    }

    for(int k = 1; k < nb_bits; k++) {
      Copy(result[k], temp[k], st_list[i%NUM_SMS]);
    }

    Constant(result[0], 0); // set lowest bit to 0

  }
}

// 11
void e_shr(Ctxt* result, Ctxt* temp, const Ctxt* a, const int shift_amt, const int nb_bits, Stream* st_list) {

  // Ctxt* temp = new Ctxt[nb_bits];

  for(int i = 0; i < nb_bits; i++) {
    Copy(result[i], a[i], st_list[i%NUM_SMS]);
  }

  for(int i = 0; i < shift_amt; i++) {
    for(int j = 0; j < nb_bits - 1; j++) {
      Copy(temp[j], result[j+1], st_list[i%NUM_SMS]);
    }


    for(int k = 0; k < nb_bits; k++) {
      Copy(result[k], temp[k], st_list[i%NUM_SMS]);
    }

    Constant(result[nb_bits - 1], 0); // set highest bit to 0

  }
}

// 12

__global__
void AddToOp(Torus* out, Torus* in0, uint32_t n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < (n+1)) {
    out[i] += in0[i];
  }
}

__global__
void SubToOp(Torus* out, Torus* in0, uint32_t n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < (n+1)) {
    out[i] -= in0[i];
  }
}

void Mux(Ctxt& result, const Ctxt& a, const Ctxt& b, const Ctxt& c, Stream st) {

  CtxtCopyH2D(a, st);
  CtxtCopyH2D(b, st);
  CtxtCopyH2D(c, st);

  static const Torus mu_boot = ModSwitchToTorus(1, 8);
  static const Torus mu_mux = ModSwitchToTorus(1, 8);
  static const Torus mu_and = ModSwitchToTorus(-1, 8);

  Ctxt* temp_result = new Ctxt;
  Ctxt* temp_result1 = new Ctxt(false, 1024);
  Ctxt* u1 = new Ctxt(false, 1024);
  Ctxt* u2 = new Ctxt(false, 1024);

  NoiselessTrivial(*temp_result, mu_and);
  AddToOp<<<1,512,0,st.st()>>>(temp_result->lwe_sample_device_->data(), a.lwe_sample_device_->data(), a.lwe_sample_device_->n());
  AddToOp<<<1,512,0,st.st()>>>(temp_result->lwe_sample_device_->data(), b.lwe_sample_device_->data(), b.lwe_sample_device_->n());
  Bootstrap_woKS(u1->lwe_sample_device_, temp_result->lwe_sample_device_, mu_boot, st.st());

  NoiselessTrivial(*temp_result, mu_and);
  SubToOp<<<1,512,0,st.st()>>>(temp_result->lwe_sample_device_->data(), a.lwe_sample_device_->data(), a.lwe_sample_device_->n());
  AddToOp<<<1,512,0,st.st()>>>(temp_result->lwe_sample_device_->data(), c.lwe_sample_device_->data(), c.lwe_sample_device_->n());
  Bootstrap_woKS(u2->lwe_sample_device_, temp_result->lwe_sample_device_, mu_boot, st.st());

  NoiselessTrivial(*temp_result1, mu_mux);
  AddToOp<<<1,512,0,st.st()>>>(temp_result1->lwe_sample_device_->data(), u1->lwe_sample_device_->data(), u1->lwe_sample_device_->n());
  AddToOp<<<1,512,0,st.st()>>>(temp_result1->lwe_sample_device_->data(), u2->lwe_sample_device_->data(), u2->lwe_sample_device_->n());
  KeySwitchHost(result.lwe_sample_device_, temp_result1->lwe_sample_device_, st.st());
  CtxtCopyD2H(result, st);
}

void e_mux(Ctxt* result, const Ctxt* sel, const Ctxt* a, const Ctxt* b, const int nb_bits, Stream* st_list) {
    for (int i=0; i<nb_bits; i++) {
        Mux(result[i], sel[0], a[i], b[i], st_list[i%NUM_SMS]);
    }
}

// Arithmetic Functions

// 13
void comparator(Ctxt* result, Ctxt* not_a, Ctxt* not_b, Ctxt* temp, Ctxt* greater_than, Ctxt* equal, Ctxt* less_than, const Ctxt* a, const Ctxt* b, const int nb_bits, const int select, Stream st) {

    // Ctxt* not_a = new Ctxt;
    // Ctxt* not_b = new Ctxt;
    // Ctxt* temp = new Ctxt[10];
    // Ctxt* greater_than = new Ctxt;
    // Ctxt* equal = new Ctxt;
    // Ctxt* less_than = new Ctxt;

    // initialize cascading inputs
    Constant(greater_than[0], 0);
    Constant(less_than[0], 0);
    Constant(equal[0], 1);

    //run 1 bit comparators
    for (int i = (nb_bits - 1); i > -1; i--) {
    //for (int i = 0; i < nb_bits; i++) {
        // invert inputs
        Not(not_a[0], a[i], st);
        Not(not_b[0], b[i], st);

        // compute greater than path
        Not(temp[0], greater_than[0], st);
        Nand(temp[1], a[i], not_b[0], st);
        Nand(temp[2], temp[1], equal[0], st);
        Not(temp[3], temp[2], st);
        Nand(greater_than[0], temp[0], temp[3], st);
        Not(temp[8], less_than[0], st);
        And(greater_than[0], temp[8], greater_than[0], st);

        // compute less than path
        Not(temp[4], less_than[0], st);
        Nand(temp[5], not_a[0], b[i], st);
        Nand(temp[6], temp[5], equal[0], st);
        Not(temp[7], temp[6], st);
        Nand(less_than[0], temp[7], temp[4], st);
        Not(temp[9], greater_than[0], st);
        And(less_than[0], temp[9], less_than[0], st);

        // compute equality path
        Nor(equal[0], greater_than[0], less_than[0], st);
    }

    // select desired output
    if (select == 0) { // ecmpeq
        Copy(result[0], equal[0], st);
    }
    else if (select == 1) { // ecmpl
        Copy(result[0], less_than[0], st);
    }
    else if (select == 2) { // ecmpg
        Copy(result[0], greater_than[0], st);
    }
    else if (select == 3) { // ecmpgeq
        Or(result[0], equal[0], greater_than[0], st);
    }
    else if (select == 4) { // ecmpleq
        Or(result[0], equal[0], less_than[0], st);
    }
    else if (select == 5) { // ecmpneq
      Copy(result[0], equal[0], st);
      Not(result[0], result[0], st);
    }

    // align to word size
    for (int i = 1; i < nb_bits; i++) {
        Constant(result[i], 0);
    }

    //delete [] not_a;
    //delete [] not_b;
    //delete [] temp;
    //delete [] greater_than;
    //delete [] equal;
    //delete [] less_than;
}

void ripple_adder(Ctxt* result, const Ctxt* a, const Ctxt* b, const int nb_bits, Stream* st_list) {

    Ctxt* carry = new Ctxt[nb_bits+1];
    Ctxt* temp = new Ctxt[1];
    //initialize first carry to 0
    Constant(carry[0], 0);

    //run full adders
    for (int i = 0; i < nb_bits; i++) {
      Xor(temp[0], a[i], b[i], st[i%NUM_SMS]);
      // Compute sum
      Xor(result[i], carry[i], temp[0], st[i%NUM_SMS]);
      // Compute carry
      Mux(carry[i+1], temp[0], carry[i], a[i], st[i%NUM_SMS]);
    }

    delete [] carry;
    delete [] temp;
}

//7
void adder(Ctxt* result, Ctxt* carry_out, Ctxt** p, Ctxt** g, Ctxt* temp, const Ctxt* a, const Ctxt* b, const int nb_bits, Stream* st_list) {

    // Stage 1
    // level 1: 32 parallel gates
    for (int i = 0; i < 16; i++) {
      Xor(p[0][i], a[i], b[i], st_list[i%NUM_SMS]);
      And(g[0][i], a[i], b[i], st_list[(16+i)%NUM_SMS]);
    }

    Synchronize();

    // Stage 2
    // level 2: 30 parallel gates
    // level 3: 15 parallel gates (OR gate depends on AND)
    for (int i = 0; i < 16; i++) {
      if (i == 0) {
        Copy(g[1][i], g[0][i], st_list[i%NUM_SMS]);
      }
      else {
        And(temp[i], p[0][i], g[0][i-1], st_list[i%NUM_SMS]);
        Or(g[1][i], temp[i], g[0][i], st_list[i%NUM_SMS]);
        And(p[1][i], p[0][i], p[0][i-1], st_list[(16+i)%NUM_SMS]);
      }
    }

    Synchronize();

    // Stage 3
    // level 4: 27 parallel gates 
    // level 5: 14 parallel gates
    for (int i = 1; i < 16; i++) {
      if (i == 1) {
        Copy(g[2][i], g[1][i], st_list[i%NUM_SMS]);
      }
      else if (i == 2) {
        And(temp[i], g[1][i-2], p[1][i], st_list[i%NUM_SMS]);
        Or(g[2][i], temp[i], g[1][i], st_list[i%NUM_SMS]);
      }
      else {
        And(temp[i], p[1][i], g[1][i-2], st_list[i%NUM_SMS]);
        Or(g[2][i], temp[i], g[1][i], st_list[i%NUM_SMS]);
        And(p[2][i], p[1][i], p[1][i-2], st_list[(16+i)%NUM_SMS]);
      }
    }

    Synchronize();

    // Stage 4
    // level 6: 21 parallel gates
    // level 7: 12 parallel gates
    for (int i = 3; i < 16; i++) {
      if (i == 3) {
        Copy(g[3][i], g[2][i], st_list[i%NUM_SMS]);
      }
      else if (i == 4) {
        And(temp[i], g[1][i-4], p[2][i], st_list[i%NUM_SMS]);
        Or(g[3][i], temp[i], g[2][i], st_list[i%NUM_SMS]);
      }
      else if (i < 7) {
        And(temp[i], g[2][i-4], p[2][i], st_list[i%NUM_SMS]);
        Or(g[3][i], temp[i], g[2][i], st_list[i%NUM_SMS]);
      }
      else {
        And(temp[i], p[2][i], g[2][i-4], st_list[i%NUM_SMS]);
        Or(g[3][i], temp[i], g[2][i], st_list[i%NUM_SMS]);
        And(p[3][i], p[2][i], p[2][i-4], st_list[(16+i)%NUM_SMS]);
      }
    }

    Synchronize();

    // Stage 5
    // level 8: 9 parallel gates
    // level 9: 8 parallel gates
    for (int i = 7; i < 16; i++) {
      if (i == 7) {
        Copy(g[4][i], g[3][i], st_list[i%NUM_SMS]);
      }
      else if (i == 8) {
        And(temp[i], g[1][i-8], p[3][i], st_list[i%NUM_SMS]);
        Or(g[4][i], temp[i], g[3][i], st_list[i%NUM_SMS]);
      }
      else if (i < 11) {
        And(temp[i], g[2][i-8], p[3][i], st_list[i%NUM_SMS]);
        Or(g[4][i], temp[i], g[3][i], st_list[i%NUM_SMS]);
      }
      else if (i < 15) {
        And(temp[i], g[3][i-8], p[3][i], st_list[i%NUM_SMS]);
        Or(g[4][i], temp[i], g[3][i], st_list[i%NUM_SMS]);
      }
      else {
        And(temp[i], p[3][i], g[3][i-8], st_list[i%NUM_SMS]);
        Or(g[4][i], temp[i], g[3][i], st_list[i%NUM_SMS]);
        And(p[4][i], p[3][i], p[3][i-8], st_list[(16+i)%NUM_SMS]);
      }
    }

    Synchronize();

    // Generate Sum and Carry Out
    // level 10: 15 parallel gates
    for (int i = 0; i < 16; i++) {
      if (i == 0) {
        Copy(result[i], p[0][i], st_list[i%NUM_SMS]);
      }
      else if (i == 1) {
        Xor(result[i], g[1][i-1], p[0][i], st_list[i%NUM_SMS]);
      }
      else if (i < 4) {
        Xor(result[i], g[2][i-1], p[0][i], st_list[i%NUM_SMS]);
      }
      else if (i < 8) {
        Xor(result[i], g[3][i-1], p[0][i], st_list[i%NUM_SMS]);
      }
      else {
        Xor(result[i], g[4][i-1], p[0][i], st_list[i%NUM_SMS]);
      }
    }
    Copy(carry_out[0], g[4][15], st_list[0]);

    return;
}

// 6
void subtracter(Ctxt* result, Ctxt* b, Ctxt* one, Ctxt* borrow_out, Ctxt** p, Ctxt** g, Ctxt* temp, const Ctxt* a, const Ctxt* b_in, const int nb_bits, Stream* st_list) {

    // compute 2's complement of b_in
    for (int i = 0; i < 16; i++) {
      Not(b[i], b_in[i], st_list[i%NUM_SMS]);
      if (i == 0) {
        Constant(one[i], 1);
      }
      else {
        Constant(one[i], 0);
      }
    }
    adder(b, borrow_out, p, g, temp, b, one, nb_bits, st_list);

    // stage 1
    for (int i = 0; i < 16; i++) {
      Xor(p[0][i], a[i], b[i], st_list[i%NUM_SMS]);
      And(g[0][i], a[i], b[i], st_list[i%NUM_SMS]);
    }

    Synchronize();

    // stage 2
    for (int i = 0; i < 16; i++) {
      if (i == 0) {
        Copy(g[1][i], g[0][i], st_list[i%NUM_SMS]);
      }
      else {
        And(temp[i], p[0][i], g[0][i-1], st_list[i%NUM_SMS]);
        Or(g[1][i], temp[i], g[0][i], st_list[i%NUM_SMS]);
        And(p[1][i], p[0][i], p[0][i-1], st_list[i%NUM_SMS]);
      }
    }

    Synchronize();

    // stage 3
    for (int i = 1; i < 16; i++) {
      if (i == 1) {
        Copy(g[2][i], g[1][i], st_list[i%NUM_SMS]);
      }
      else if (i == 2) {
        And(temp[i], g[1][i-2], p[1][i], st_list[i%NUM_SMS]);
        Or(g[2][i], temp[i], g[1][i], st_list[i%NUM_SMS]);
      }
      else {
        And(temp[i], p[1][i], g[1][i-2], st_list[i%NUM_SMS]);
        Or(g[2][i], temp[i], g[1][i], st_list[i%NUM_SMS]);
        And(p[2][i], p[1][i], p[1][i-2], st_list[i%NUM_SMS]);
      }
    }

    Synchronize();

    // stage 4
    for (int i = 3; i < 16; i++) {
      if (i == 3) {
        Copy(g[3][i], g[2][i], st_list[i%NUM_SMS]);
      }
      else if (i == 4) {
        And(temp[i], g[1][i-4], p[2][i], st_list[i%NUM_SMS]);
        Or(g[3][i], temp[i], g[2][i], st_list[i%NUM_SMS]);
      }
      else if (i < 7) {
        And(temp[i], g[2][i-4], p[2][i], st_list[i%NUM_SMS]);
        Or(g[3][i], temp[i], g[2][i], st_list[i%NUM_SMS]);
      }
      else {
        And(temp[i], p[2][i], g[2][i-4], st_list[i%NUM_SMS]);
        Or(g[3][i], temp[i], g[2][i], st_list[i%NUM_SMS]);
        And(p[3][i], p[2][i], p[2][i-4], st_list[i%NUM_SMS]);
      }
    }

    Synchronize();

    // stage 5
    for (int i = 7; i < 16; i++) {
      if (i == 7) {
        Copy(g[4][i], g[3][i], st_list[i%NUM_SMS]);
      }
      else if (i == 8) {
        And(temp[i], g[1][i-8], p[3][i], st_list[i%NUM_SMS]);
        Or(g[4][i], temp[i], g[3][i], st_list[i%NUM_SMS]);
      }
      else if (i < 11) {
        And(temp[i], g[2][i-8], p[3][i], st_list[i%NUM_SMS]);
        Or(g[4][i], temp[i], g[3][i], st_list[i%NUM_SMS]);
      }
      else if (i < 15) {
        And(temp[i], g[3][i-8], p[3][i], st_list[i%NUM_SMS]);
        Or(g[4][i], temp[i], g[3][i], st_list[i%NUM_SMS]);
      }
      else {
        And(temp[i], p[3][i], g[3][i-8], st_list[i%NUM_SMS]);
        Or(g[4][i], temp[i], g[3][i], st_list[i%NUM_SMS]);
        And(p[4][i], p[3][i], p[3][i-8], st_list[i%NUM_SMS]);
      }
    }

    Synchronize();

    // Generate Sum and Carry Out
    for (int i = 0; i < 16; i++) {
      if (i == 0) {
        Copy(result[i], p[0][i], st_list[i%NUM_SMS]);
      }
      else if (i == 1) {
        Xor(result[i], g[1][i-1], p[0][i], st_list[i%NUM_SMS]);
      }
      else if (i < 4) {
        Xor(result[i], g[2][i-1], p[0][i], st_list[i%NUM_SMS]);
      }
      else if (i < 8) {
        Xor(result[i], g[3][i-1], p[0][i], st_list[i%NUM_SMS]);
      }
      else {
        Xor(result[i], g[4][i-1], p[0][i], st_list[i%NUM_SMS]);
      }
    }
    Copy(borrow_out[0], g[4][15], st_list[0]);

    return;
}

// 8
void multiplier(Ctxt* result, Ctxt* overflow, Ctxt** prods, Ctxt** p, Ctxt** g, Ctxt* temp, Ctxt* temp_shift, const Ctxt* a, const Ctxt* b, const int nb_bits, Stream* st_list) {

    // Ctxt* prods[nb_bits];
    // for (int i = 0; i < nb_bits; i++) {
    //   prods[i] = new Ctxt[nb_bits];
    // }

    // generate partial products
    for (int i = 0; i < nb_bits; i++) {
      e_shl(prods[i], temp_shift, a, i, nb_bits, st_list);
      for (int j = 0; j < nb_bits; j++) {
        And(prods[i][j], prods[i][j], b[i], st_list[i%NUM_SMS]);
      }
    }

    // load first partial product into result
    for (int i = 0; i < nb_bits; i++) {
      Copy(result[i], prods[0][i], st_list[i%NUM_SMS]);
    }

    // accumulate partial sums (no parallelization here since all 16 SMs are used by the adder)
    for (int i = 1; i < nb_bits; i++) {
      adder(result, overflow, p, g, temp, result, prods[i], nb_bits, st_list);
    }

    return;
}

void op_select(char* instruction, Stream* st_list, Ctxt* ciphertext1, Ctxt* ciphertext2, Ctxt* ciphertext3, Ctxt* result, Ctxt* old_status, Ctxt* new_status, Ctxt** prods, Ctxt** p, Ctxt** g, Ctxt* temp_adder, Ctxt* not_a, Ctxt* not_b, Ctxt* temp_comp, Ctxt* greater_than, Ctxt* equal, Ctxt* less_than, Ctxt* temp_shift, Ctxt* temp_inv, Ctxt* one) {
  char* token;
  token = strtok(instruction, " ");
  int wordSize = atoi(token);
  token = strtok(NULL, " ");
  int operation = atoi(token);

  // Ctxt* ciphertext1 = new Ctxt[wordSize];
  // Ctxt* ciphertext2 = new Ctxt[wordSize];
  // Ctxt* ciphertext3 = new Ctxt[wordSize];
  // Ctxt* old_status = new Ctxt[1];
  // Ctxt* new_status = new Ctxt[1];

  ifstream status_data ("encrypted_data/status.data");
  ReadCtxtFromFile(old_status[0], status_data);

  // Ctxt* result = new Ctxt[wordSize];

  if (operation < 9) { // 2 input ciphertexts
      token = strtok(NULL, " ");
      const std::string temp = string(token);
      ifstream ctxt_one_data (temp);
      token = strtok(NULL, " ");
      const std::string temp2 = string(token);
      ifstream ctxt_two_data (temp2);

      for (int i=0; i<wordSize; i++) {
          ReadCtxtFromFile(ciphertext1[i], ctxt_one_data);
          ReadCtxtFromFile(ciphertext2[i], ctxt_two_data);
      }


      ctxt_one_data.close();
      ctxt_two_data.close();

      if (operation == 0) { // AND
        e_and(result, ciphertext1, ciphertext2, wordSize, st_list);
      }
      else if (operation == 1) { // NAND
        e_nand(result, ciphertext1, ciphertext2, wordSize, st_list);
      }
      else if (operation == 2) { // OR
        e_or(result, ciphertext1, ciphertext2, wordSize, st_list);
      }
      else if (operation == 3) { // NOR
        e_nor(result, ciphertext1, ciphertext2, wordSize, st_list);
      }
      else if (operation == 4) { // XOR
        e_xor(result, ciphertext1, ciphertext2, wordSize, st_list);
      }
      else if (operation == 5) { // XNOR
        e_xnor(result, ciphertext1, ciphertext2, wordSize, st_list);
      }
      else if (operation == 6) { // subtract
        subtracter(result, temp_inv, one, new_status, p, g, temp_adder, ciphertext1, ciphertext2, wordSize, st_list);
        st_ctr++;
      }
      else if (operation == 7) { // add
        adder(result, new_status, p, g, temp_adder, ciphertext1, ciphertext2, wordSize, st_list);
      }
      else if (operation == 8) { // multiply
        multiplier(result, new_status, prods, p, g, temp_adder, temp_shift, ciphertext1, ciphertext2, wordSize, st_list);
        st_ctr++;
      }

  }

  else if (operation < 12) { // 1 input ciphertext
      token = strtok(NULL, " ");
      const std::string temp = string(token);
      ifstream ctxt_one_data (temp);

      for (int i=0; i<wordSize; i++) {
        ReadCtxtFromFile(ciphertext1[i], ctxt_one_data);
      }

      ctxt_one_data.close();

      if (operation == 9) { // NOT
        e_not(result, ciphertext1, wordSize, st_list);
      }
      else if (operation == 10) { // shift left
        token = strtok(NULL, " ");
        int shift_amount = atoi(token);
        e_shl(result, temp_shift, ciphertext1, shift_amount, wordSize, st_list);
      }
      else if (operation == 11) { // shift right
        token = strtok(NULL, " ");
        int shift_amount = atoi(token);
        e_shr(result, temp_shift, ciphertext1, shift_amount, wordSize, st_list);
      }
  }

  else if (operation == 12) { // MUX
      token = strtok(NULL, " ");
      const std::string temp = string(token);
      ifstream ctxt_one_data (temp);
      token = strtok(NULL, " ");
      const std::string temp2 = string(token);
      ifstream ctxt_two_data (temp2);
      token = strtok(NULL, " ");
      const std::string temp3 = string(token);
      ifstream ctxt_three_data (temp3);


      for (int i=0; i<wordSize; i++) {
          ReadCtxtFromFile(ciphertext1[i], ctxt_one_data);
          ReadCtxtFromFile(ciphertext2[i], ctxt_two_data);
          ReadCtxtFromFile(ciphertext3[i], ctxt_three_data);
      }

      ctxt_one_data.close();
      ctxt_two_data.close();
      ctxt_three_data.close();

      e_mux(result, ciphertext1, ciphertext2, ciphertext3, wordSize, st_list);
  }

  else if (operation == 13) { // comp
    token = strtok(NULL, " ");
    const std::string temp = string(token);
    ifstream ctxt_one_data (temp);
    token = strtok(NULL, " ");
    const std::string temp2 = string(token);
    ifstream ctxt_two_data (temp2);
    token = strtok(NULL, " ");
    int select = atoi(token);

    for (int i=0; i<wordSize; i++) {
        ReadCtxtFromFile(ciphertext1[i], ctxt_one_data);
        ReadCtxtFromFile(ciphertext2[i], ctxt_two_data);
    }

    ctxt_one_data.close();
    ctxt_two_data.close();

    comparator(result, not_a, not_b, temp_comp, greater_than, equal, less_than, ciphertext1, ciphertext2, wordSize, select, st_list[st_ctr%NUM_SMS]);
  }

  else if (operation == 14) { // econst
    token = strtok(NULL, " ");
    int64_t ptxt_val = atoi(token);
    for (int i=0; i<wordSize; i++) {
        Constant(result[i], (ptxt_val>>i)&1);
    }
  }
  Synchronize();
  CuCheckError();
  string fileName = gen_filename();
  for (int i=0; i<wordSize; i++)
      WriteCtxtToFile(result[i], fileName);

  if ((operation == 7) || (operation == 8)) {
    Or(new_status[0], new_status[0], old_status[0], st_list[0]);
    std::ofstream ofs;
    ofs.open("encrypted_data/status.data", std::ofstream::out | std::ofstream::trunc);
    ofs.close();
    WriteCtxtToFile(new_status[0], "encrypted_data/status.data");
  }
  // export ciphertext filename to ctxtmem.txt
  ofstream ctxt_mem;
  ctxt_mem.open("ctxtMem.txt", ios_base::app);
  ctxt_mem << fileName;
  ctxt_mem << "\n";
  ctxt_mem.close();
}

int listen_for_inst(int sockfd, Stream* st_list, Ctxt* ciphertext1, Ctxt* ciphertext2, Ctxt* ciphertext3, Ctxt* result, Ctxt* old_status, Ctxt* new_status, Ctxt** prods, Ctxt** p, Ctxt** g, Ctxt* temp, Ctxt* not_a, Ctxt* not_b, Ctxt* temp_comp, Ctxt* greater_than, Ctxt* equal, Ctxt* less_than, Ctxt* temp_shift, Ctxt* temp_inv, Ctxt* one) {

    char buf[MAX];
    for (;;) {
        bzero(buf, MAX);

        read(sockfd, buf, sizeof(buf));
        printf("Received: %s\n", buf);
        if (strncmp("exit", buf, 4) == 0) {
            printf("Server Exit...\n");
            bzero(buf, MAX);
            buf[0] = 'O';
            buf[1] = 'K';
            write(sockfd, buf, sizeof(buf));
            return 0;
        }
        else if (buf[0] == 0) {
          printf("Erroneous message, returning...\n");
          return -1;
        }
        op_select(buf, st_list, ciphertext1, ciphertext2, ciphertext3, result, old_status, new_status, prods, p, g, temp, not_a, not_b, temp_comp, greater_than, equal, less_than, temp_shift, temp_inv, one);
        bzero(buf, MAX);
        buf[0] = 'O';
        buf[1] = 'K';
        printf("Sending OK...\n");
        int error_check = write(sockfd, buf, sizeof(buf));
        if (error_check < 0) {
          return -1;
        }
    }
}

int main(int argc, char** argv) {

    hipSetDevice(0);

    // read evaluation key from file
    PubKey bk;
    ReadPubKeyFromFile(bk, "eval.key");
    Initialize(bk);

    // Allocate ctxt objects up front
    Ctxt* ciphertext1 = new Ctxt[16];
    Ctxt* ciphertext2 = new Ctxt[16];
    Ctxt* ciphertext3 = new Ctxt[16];
    Ctxt* result = new Ctxt[16];
    Ctxt* old_status = new Ctxt[1];
    Ctxt* new_status = new Ctxt[1];
    Ctxt* prods[16];
    for (int i = 0; i < 16; i++) {
      prods[i] = new Ctxt[16];
    }
    Ctxt* p[5];
    Ctxt* g[5];
    for (int i = 0; i < (5); i++) {
      p[i] = new Ctxt[16];
      g[i] = new Ctxt[16];
    }
    Ctxt* temp = new Ctxt[16];
    Ctxt* not_a = new Ctxt;
    Ctxt* not_b = new Ctxt;
    Ctxt* temp_comp = new Ctxt[10];
    Ctxt* greater_than = new Ctxt;
    Ctxt* equal = new Ctxt;
    Ctxt* less_than = new Ctxt;
    Ctxt* temp_shift = new Ctxt[16];
    Ctxt* temp_inv = new Ctxt[16];
    Ctxt* one = new Ctxt[16];


    // create streams
    Stream* st_list = new Stream[NUM_SMS];
    for (int i = 0; i < NUM_SMS; i++) {
      st_list[i].Create();
    }

    int sockfd, connfd;
    socklen_t len;
    int error_check = -1;
    struct sockaddr_in servaddr, cli;
    while (error_check < 0) {
      // socket create and verification
      sockfd = socket(AF_INET, SOCK_STREAM, 0);
      if (sockfd == -1) {
          printf("socket creation failed...\n");
          exit(0);
      }
      else
          printf("Socket successfully created..\n");
      bzero(&servaddr, sizeof(servaddr));

      // assign IP, PORT
      servaddr.sin_family = AF_INET;
      servaddr.sin_addr.s_addr = htonl(INADDR_ANY);
      servaddr.sin_port = htons(PORT);

      // Binding newly created socket to given IP and verification
      if ((bind(sockfd, (SA*)&servaddr, sizeof(servaddr))) != 0) {
          printf("socket bind failed...\n");
          exit(0);
      }
      else
          printf("Socket successfully binded..\n");

      // Now server is ready to listen and verification
      if ((listen(sockfd, 5)) != 0) {
          printf("Listen failed...\n");
          exit(0);
      }
      else
          printf("Server listening..\n");
      len = sizeof(cli);

      // Accept the data packet from client and verification
      connfd = accept(sockfd, (SA*)&cli, &len);
      if (connfd < 0) {
          printf("Server accept failed...\n");
          exit(0);
      }
      else
          printf("Server accepted the client...\n");

      // Function for chatting between client and server
      error_check = listen_for_inst(connfd, st_list, ciphertext1, ciphertext2, ciphertext3, result, old_status, new_status, prods, p, g, temp, not_a, not_b, temp_comp, greater_than, equal, less_than, temp_shift, temp_inv, one);

      // After chatting close the socket
      close(sockfd);
    }
    //clean up all pointers
    CleanUp();

    delete [] ciphertext1;
    delete [] ciphertext2;
    delete [] ciphertext3;
    delete [] result;
    delete [] old_status;
    delete [] new_status;
    for (int size_i = 0; size_i < 16; size_i++) {
      delete [] prods[size_i];
    }
    for (int size_i = 0; size_i < 5; size_i++) {
      delete [] p[size_i];
      delete [] g[size_i];
    }
    delete [] temp;
    delete [] not_a;
    delete [] not_b;
    delete [] temp_comp;
    delete greater_than;
    delete equal;
    delete less_than;
    delete [] temp_shift;
    delete [] temp_inv;
    delete [] one;
}
